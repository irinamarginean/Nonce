﻿
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <memory.h>
#include <iostream>
#include <string>
#include <sstream>
#include <iomanip>
#include <bitset>
#include <vector>

using namespace std;

#define SHA_BLOCK_SIZE 20
#define INPUT "ab"
#define INPUT_SIZE 2

typedef unsigned char BYTE;
typedef unsigned int WORD;
typedef unsigned long long LONG;

const BYTE HASH_VALUE[] = {0xb7, 0x3f, 0x22, 0xd7, 0x93, 0x61, 0xce, 0xa9, 0x15, 0x1c, 0x9f, 0x7c, 0x30, 0x99, 0x3f, 0x54, 0xe6, 0xb4, 0xed, 0x1a};
const string HASH_VALUE_STRING = "b73f22d79361cea9151c9f7c30993f54e6b4ed1a";  

typedef struct {
	BYTE data[64];
	WORD datalen;
	LONG bitlen;
	WORD state[5];
	WORD k[4];
} CUDA_SHA1_CTX;

#ifndef ROTLEFT
#define ROTLEFT(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
#endif

__device__ void cuda_sha1_init(CUDA_SHA1_CTX* dtx) 
{
	dtx->datalen = 0;
	dtx->bitlen = 0;
	dtx->state[0] = 0x67DE2A01;
	dtx->state[1] = 0xBB03E28C;
	dtx->state[2] = 0x011EF1DC;
	dtx->state[3] = 0x9293E9E2;
	dtx->state[4] = 0xCDEF23A9;
	dtx->k[0] = 0x5a827999;
	dtx->k[1] = 0x6ed9eba1;
	dtx->k[2] = 0x8f1bbcdc;
	dtx->k[3] = 0xca62c1d6;
}

__device__ void cuda_sha1_transform(CUDA_SHA1_CTX* dtx, const BYTE data[])
{
	WORD A, B, C, D, E, W[80], temp;

	for (int i = 0, j = 0; i < 16; i++, j += 4)
	{
		W[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + data[j + 3];
	}

	for (int i = 16; i < 80; i++)
	{
		W[i] = W[i - 3] ^ W[i - 8] ^ W[i - 14] ^ W[i - 16];
		W[i] = (W[i] << 1) | (W[i] >> 31);
	}

	A = dtx->state[0];
	B = dtx->state[1];
	C = dtx->state[2];
	D = dtx->state[3];
	E = dtx->state[4];

	for (int i = 0; i < 20; i++)
	{
		temp = ROTLEFT(A, 5) + ((B ^ C) & (~B & D)) + E + W[i] + dtx->k[0];
		E = D;
		D = C;
		C = ROTLEFT(B, 30);
		B = A;
		A = temp;
	}

	for (int i = 20; i < 40; i++)
	{
		temp = ROTLEFT(A, 5) + (B ^ C ^D) + E + W[i] + dtx->k[1];
		E = D;
		D = C;
		C = ROTLEFT(B, 30);
		B = A;
		A = temp;
	}

	for (int i = 40; i < 60; i++)
	{
		temp = ROTLEFT(A, 5) + ((B ^ C) & (B ^ D) & (C ^ D)) + E + W[i] + dtx->k[2];
		E = D;
		D = C;
		C = ROTLEFT(B, 30);
		B = A;
		A = temp;
	}

	for (int i = 60; i < 80; i++)
	{
		temp = ROTLEFT(A, 5) + (B ^ C ^D) + E + W[i] + dtx->k[3];
		E = D;
		D = C;
		C = ROTLEFT(B, 30);
		B = A;
		A = temp;
	}

	dtx->state[0] += A;
	dtx->state[1] += B;
	dtx->state[2] += C;
	dtx->state[3] += D;
	dtx->state[4] += E;
}

__device__ void cuda_sha1_update(CUDA_SHA1_CTX* dtx, const BYTE data[], LONG length)
{
	for (LONG i = 0; i < length; i++)
	{
		dtx->data[dtx->datalen] = data[i];
		dtx->datalen++;
		if (dtx->datalen == 64)
		{
			cuda_sha1_transform(dtx, dtx->data);
			dtx->bitlen += 512;
			dtx->datalen = 0;
		}
	}
}

__device__ void cuda_sha1_final(CUDA_SHA1_CTX *dtx, BYTE hash[])
{
	WORD i = dtx->datalen;

	if (dtx->datalen < 56)
	{
		dtx->data[i++] = 0x80;
		while (i < 56)	
		{
			dtx->data[i++] = 0x00;
		}
	}
	else 
	{
		dtx->data[i++] = 0x80;
		while (i < 64)
		{
			dtx->data[i++] = 0x00;
		}
		cuda_sha1_transform(dtx, dtx->data);
		memset(dtx->data, 0, 56);
	}

	dtx->bitlen += dtx->datalen * 8;
	dtx->data[63] = dtx->bitlen;
	dtx->data[62] = dtx->bitlen >> 8;
	dtx->data[61] = dtx->bitlen >> 16;
	dtx->data[60] = dtx->bitlen >> 24;
	dtx->data[59] = dtx->bitlen >> 32;
	dtx->data[58] = dtx->bitlen >> 40;
	dtx->data[57] = dtx->bitlen >> 48;
	dtx->data[56] = dtx->bitlen >> 56;
	cuda_sha1_transform(dtx, dtx->data);

	for (i = 0; i < 4; ++i) {
		hash[i]      = (dtx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (dtx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (dtx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (dtx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (dtx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

 __device__ BYTE* convert_long_to_byte(LONG number, BYTE* string, int* size)
{
	int i = 0;

	if (number == 0) {
		string[i++] = '0';
		return string;
	}

	while (number != 0) {
		int remainder = number % 10;
		string[i++] = (remainder > 9) ? (remainder - 10) + 'a' : remainder + '0';
		number = number / 10;
	}

	// reverse the string
	int begin = 0, end = *size - 1;
	while (begin < end) {
		char begin_char = *(string + begin);
		char end_char = *(string + end);
		*(string + begin) = end_char;
		*(string + end) = begin_char;
		begin++;
		end--;
	}

	*size = i;

	return string;
}

__global__ void kernel_sha1_nonce_hash(LONG* result, BYTE* hash, bool* is_found, LONG* n, BYTE* original_hash_value) 
{
	CUDA_SHA1_CTX dtx;
	BYTE output[SHA_BLOCK_SIZE];
	int nonce_size = 0;
	BYTE nonce[SHA_BLOCK_SIZE];
	BYTE hash_input[SHA_BLOCK_SIZE + INPUT_SIZE];

	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	LONG nonce_input = tid + *n;

	//printf("nonce: %ld \n ", nonce_input);

	convert_long_to_byte(nonce_input, nonce, &nonce_size);

	memcpy(hash_input, (BYTE*) INPUT, INPUT_SIZE);
	memcpy(hash_input + INPUT_SIZE, nonce, nonce_size);
	memset(output, 0x0, SHA_BLOCK_SIZE);

	cuda_sha1_init(&dtx);
	cuda_sha1_update(&dtx, hash_input, INPUT_SIZE + nonce_size);
	cuda_sha1_final(&dtx, output);

	bool are_equal = true;

	for (int index = 0; index < SHA_BLOCK_SIZE; index++)
	{
		if (output[index] != original_hash_value[index])
		{
			are_equal = false;
		}
	}

	if (are_equal) 
	{
		*is_found = true;
		int i = 0;
		do 
		{
			hash[i] = output[i];
		} 
		while (output[i++] != 0);
		*result = nonce_input;
	}
}

int main()
{
	int grid_size = 256, block_size = 256;
	LONG nonce_size = sizeof(LONG);
	LONG thread_count = 0, step = 0;
	bool h_is_nonce_found = false;

	/*cudaDeviceProp device_prop;
	cudaGetDeviceProperties(&device_prop, 0);
	cudaOccupancyMaxPotentialBlockSize(&grid_size, &block_size, kernel_sha1_nonce_hash);*/

	grid_size = 256, block_size = 256;

	thread_count = grid_size * block_size;

	BYTE original_hash[SHA_BLOCK_SIZE];
	copy(begin(HASH_VALUE), end(HASH_VALUE), begin(original_hash));

	// CPU vars
	
	LONG h_found_nonce = 0;
	BYTE* h_computed_hash = (BYTE*) malloc(SHA_BLOCK_SIZE);

	memset(h_computed_hash, 0, SHA_BLOCK_SIZE);

	// CUDA vars

	LONG* d_found_nonce;
	BYTE* d_computed_hash;
	bool* d_is_nonce_found;
	BYTE* d_original_hash;
	LONG* d_step;

	hipMalloc((void**)&d_found_nonce, sizeof(LONG));
	hipMalloc((void**)&d_step, sizeof(LONG));
	hipMalloc((void**)&d_computed_hash, SHA_BLOCK_SIZE);
	hipMalloc((void**)&d_original_hash, SHA_BLOCK_SIZE);
	hipMalloc((void**)&d_is_nonce_found, sizeof(bool));
	
	hipMemcpy(d_is_nonce_found, &h_is_nonce_found, sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(d_original_hash, &original_hash, SHA_BLOCK_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_step, &step, sizeof(LONG), hipMemcpyHostToDevice);

	string hexString;
	string hexString_nonce;

	step = 0;

	while (!h_is_nonce_found) 
	{
		kernel_sha1_nonce_hash<<<grid_size, block_size>>>(d_found_nonce, d_computed_hash, d_is_nonce_found, d_step, d_original_hash);

		hipDeviceSynchronize();

		hipMemcpy(&h_is_nonce_found, d_is_nonce_found, sizeof(bool), hipMemcpyDeviceToHost);

		step += thread_count;

		hipMemcpy(d_step, &step, sizeof(LONG), hipMemcpyHostToDevice);
	}

	hipMemcpy(h_computed_hash, d_computed_hash, SHA_BLOCK_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(&h_found_nonce, d_found_nonce, nonce_size, hipMemcpyDeviceToHost);

	stringstream ss_nonce;
	ss_nonce << "0x" << setw(8) << setfill('0') << hex << h_found_nonce;
	string nonce_hex_string = ss_nonce.str();
		
	stringstream ss_hash;
	for (int i = 0; i < SHA_BLOCK_SIZE; i++) {
		ss_hash << std::hex << std::setw(2) << std::setfill('0') << static_cast<int>(h_computed_hash[i]);
	}

	hexString = ss_hash.str();

	cout << "Original hash: " << HASH_VALUE_STRING << endl;
	cout << "Computed hash: " << hexString << endl;

	cout << "hash in int: ";

	for (int i = 0; i < SHA_BLOCK_SIZE; i++)
	{
		cout << (int)h_computed_hash[i] << " ";
	}

	cout << endl << "===================================> NONCE: " << nonce_hex_string << endl;

	free(h_computed_hash);
	hipFree(d_found_nonce);
	hipFree(d_computed_hash);
	hipFree(d_is_nonce_found);
	hipFree(d_original_hash);
	hipFree(d_step);
}
